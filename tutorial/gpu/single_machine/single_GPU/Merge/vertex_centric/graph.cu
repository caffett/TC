#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>

#include "comm.h"
#include "graph.h"
#include "wtime.h"

using namespace std;

/*******************GPU Functions***********************/
// 1 thread <-> 1 vertex -> imbalance
__global__ void vertex_merge_kernel(	
	vertex_t*	adj,
	index_t*	begin,
	index_t		Ns,	// The start thread id
	index_t		Ne,	// The end thread id
	index_t*	count
){
	__shared__ index_t local_count[THDS_NUM];
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x);
	index_t thd_count=0;

	while(tid<Ne){
		vertex_t A = tid;
		index_t degree_A = begin[A+1]-begin[A];
		vertex_t* a = &(adj[begin[A]]);

		for(int i=0; i<degree_A; i++){
			vertex_t B = adj[begin[A]+i];
			index_t degree_B = begin[B+1]-begin[B];
			vertex_t* b = &(adj[begin[B]]);

			index_t index_A  = 0;
			index_t index_B  = 0;
			index_t x,y;
			while(index_A<degree_A && index_B<degree_B){
				x=a[index_A];
				y=b[index_B];

				if(x<y){
					index_A++;
				}
				else if(x>y){
					index_B++;
				}
				else if(x==y){
					index_A++;
					index_B++;
					thd_count++;
				}
			}
		}
		tid += gridDim.x*blockDim.x;
	}
	//reduce
	local_count[threadIdx.x] = thd_count;
	__syncthreads();

	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val += local_count[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();
}

__global__ void reduce_kernel(index_t* count)
{
	index_t val = 0;
	for(int i=0; i<BLKS_NUM; i++){
		val += count[i];
	}
	count[0] = val;
}

/*******************CPU Functions***********************/
void printGraph(vertex_t vertCount, 
		vertex_t* head, 
		vertex_t* adj, 
		index_t* begin){
	for(vertex_t i=0; i<vertCount; i++){
		if(begin[i+1]>begin[i]){
			cout<<begin[i]<<" "<<begin[i+1]-begin[i]<<": ";
		}
//		for(int j=0; j<degree[i]; j++){
		for(vertex_t j=0; j<begin[i+1]-begin[i]; j++){
			cout<<head[begin[i]+j]<<"-"<<adj[begin[i]+j]<<" ";
		}
		if(begin[i+1]>begin[i]){
			cout<<"\n";
		}
	}
}


void graph::scan(){
	// Select the GPU to excute kernel
	hipSetDevice(1);

	// The variables of GPU
	index_t* dev_begin;
	vertex_t* dev_adj_list;
	index_t* dev_count;

	H_ERR(hipMalloc((void**)&dev_begin, (vertex_count+1)*sizeof(index_t)));
	H_ERR(hipMalloc((void**)&dev_adj_list, RankedEdgeCount*sizeof(vertex_t)));
	H_ERR(hipMalloc((void**)&dev_count, BLKS_NUM*sizeof(index_t)));

	// Be careful, all the list have been ranked.
	H_ERR(hipMemcpy(dev_adj_list, RankedAdj, RankedEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice));
	H_ERR(hipMemcpy(dev_begin, RankedBegin, (vertex_count+1)*sizeof(index_t), hipMemcpyHostToDevice));

	double t0 = wtime();
	// call GPU merge based function kernel
	vertex_merge_kernel<<<BLKS_NUM,THDS_NUM>>>
	(	
		dev_adj_list,
		dev_begin,
		0,
		vertex_count,
		dev_count
	);
	H_ERR(hipDeviceSynchronize());

	// reduce the results
	reduce_kernel <<<1,1>>>(dev_count);
	int result;
	H_ERR(hipMemcpy(&result, dev_count, sizeof(int), hipMemcpyDeviceToHost));
	double t1 = wtime();
	
	cout << "Total count: " << result << endl;
	cout << "Time consume: " << t1-t0 << " s" << endl;

	H_ERR(hipFree(dev_count));
	H_ERR(hipFree(dev_adj_list));
	H_ERR(hipFree(dev_begin));
}


graph::graph(string jsonfile){
	cout<<"read from folder "<<jsonfile<<endl;
	
	string s_begin = jsonfile+"/begin.bin";
	string s_adj = jsonfile+"/adjacent.bin";
	string s_head = jsonfile+"/head.bin";
	string s_degree = jsonfile+"/degree.bin";

	char* begin_file = const_cast<char*>(s_begin.c_str());
	char* adj_file = const_cast<char*>(s_adj.c_str());
	char* head_file = const_cast<char*>(s_head.c_str());
	char* degree_file = const_cast<char*>(s_degree.c_str());

	vertex_count = fsize(begin_file)/sizeof(index_t) - 1;
	edge_count = fsize(head_file)/sizeof(vertex_t);

	FILE *pFile= fopen(adj_file,"rb");
	adj_list = (vertex_t *)malloc(fsize(adj_file));
	fread(adj_list,sizeof(vertex_t),edge_count,pFile);
	fclose(pFile);

	FILE *pFile1= fopen(head_file,"rb");
	head_list = (vertex_t *)malloc(fsize(head_file));
	fread(head_list,sizeof(vertex_t),edge_count,pFile1);
	fclose(pFile1);


	FILE *pFile3 = fopen(begin_file,"rb");
	beg_pos = (index_t *)malloc(fsize(begin_file));
	fread(beg_pos,sizeof(index_t),vertex_count+1,pFile3);
	fclose(pFile3);

	count = (index_t *)malloc(THDS_NUM*BLKS_NUM*sizeof(index_t));

	cout<<"vert:" << vertex_count<<"  edge: "<<edge_count<<endl;
}

void quickSort(vertex_t* arr, index_t left, index_t right)
{
      index_t i = left, j = right;
      vertex_t tmp;
      vertex_t pivot = arr[(left + right) / 2];
 
      /* partition */
      while (i <= j) {
            while (arr[i] < pivot)
                  i++;
            while (arr[j] > pivot)
                  j--;
            if (i <= j) {
                  tmp = arr[i];
                  arr[i] = arr[j];
                  arr[j] = tmp;
                  i++;
                  j--;
            }
      };
 
      /* recursion */
      if (left < j)
            quickSort(arr, left, j);
      if (i < right)
            quickSort(arr, i, right);
}


void graph::sort_ranked_adj(){
	// sort the adj_list
	for (vertex_t i = 0; i < vertex_count; i++){
		index_t begin = RankedBegin[i];
		index_t end = RankedBegin[i+1]-1;
		quickSort(RankedAdj, begin, end);
	}
}


void graph::rank_by_degree(){
	RankedBegin = new index_t[vertex_count+1];
	RankedBegin[0] = 0;

	// Calculate ranked begining position
	#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for (vertex_t i = 0; i < vertex_count; i++){
		RankedBegin[i+1] = 0;
		//Calculte the degree of head vertex
		vertex_t h = head_list[beg_pos[i]];
		index_t dh = beg_pos[h+1]-beg_pos[h];
		for (index_t j = beg_pos[i]; j < beg_pos[i+1]; j++){
			//Calculte the degree of adj vertex
			vertex_t a = adj_list[j];
			index_t da = beg_pos[a+1] - beg_pos[a];
			if(dh<da || (dh==da && h<a)) RankedBegin[i+1]++;
		}
	}

	// If we put this step in the above loop, we cannot make sure sycn.
	for (int i = 0; i < vertex_count; i++){
		RankedBegin[i+1] += RankedBegin[i];	
	}

	// Calculate the RankedHead and RankedAdj
	RankedEdgeCount = RankedBegin[vertex_count];
	cout <<"RankedEdgeCount: "<< RankedEdgeCount << endl;
	RankedHead = new vertex_t[RankedEdgeCount];
	RankedAdj = new vertex_t[RankedEdgeCount];

	#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for (vertex_t i = 0; i < vertex_count; i++){
		vertex_t h = head_list[beg_pos[i]];
		int dh = beg_pos[h+1]-beg_pos[h];
		int k = RankedBegin[i];

		for (index_t j = beg_pos[i]; j < beg_pos[i+1]; j++){
			vertex_t a = adj_list[j];
			index_t da = beg_pos[a+1] - beg_pos[a];
			if(dh < da || (dh == da && h<a)){
				RankedHead[k] = head_list[j];
				RankedAdj[k] = adj_list[j];
				k++;
			}
		}
	}

}


graph::~graph(){
	delete[] adj_list;
	delete[] head_list;
	delete[] beg_pos;
	delete[] RankedAdj;
	delete[] RankedHead;
	delete[] RankedBegin;
}

