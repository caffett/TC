#include "hip/hip_runtime.h"
#include <iostream>

#include "comm.h"
#include "graph.h"
#include "wtime.h"

using namespace std;

int main(int argc, char* argv[]){
	double t0, t1, total_time = 0;
	const int round = 3;

	if(argc != 2){
		perror("Please run the code with one peremeter, which is the directary path of dataset");
		return -1;
	}
	// ? G must be graph*, cannot be graph, or when hipSetDevice, it will corrupt
	graph* G = new graph(argv[1]);

	//Rank by degree
	t0 = wtime();
	G->rank_by_degree();
	t1 = wtime();
	cout << "Time of rank by degree: " << t1 - t0 << " s" << endl;

	//Sort ranked adj_list
	t0 = wtime();
	G->sort_ranked_adj();
	t1 = wtime();
	cout << "Time of sort ranked adj: " << t1 - t0 << " s" << endl;

	//Calculate triangles round times to get avarage time.
	for (int i = 0; i < round; i++){
		t0 = wtime();
		G->scan();
		t1 = wtime();
		cout << "Round " << i+1 << ": Time of scan: " << t1 - t0 << "s" << endl;
		total_time += t1-t0;
	}

	cout << "Avarage time of " << round << " round(s) :" <<  total_time/round << endl;

	return 0;
}
