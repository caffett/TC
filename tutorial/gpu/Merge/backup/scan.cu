#include "hip/hip_runtime.h"
//scan.cu
//#include "kernel.cu"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 
using namespace std;

	
__global__ void vertex_merge_kernel
(	
	vertex_t*	adj,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	__shared__ index_t local[max_thd];
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x);
	long int mycount=0;
	while(tid<Ne){
		
		vertex_t A = tid;
		index_t m = begin[A+1]-begin[A];
		vertex_t* a = &(adj[begin[A]]);
		for(int i=0; i<m; i++){
			vertex_t B = adj[begin[A]+i];
			index_t n = begin[B+1]-begin[B];//degree[B];

			vertex_t* b = &(adj[begin[B]]);
			

			vertex_t lowA  = 0;
			vertex_t lowB  = 0;
			vertex_t highA = m;
			vertex_t highB = n;
			vertex_t x,y;
			while(lowA<highA && lowB<highB){
				x=a[lowA];
				y=b[lowB];
				if(x<y){
					lowA++;
				}
				else if(x>y){
					lowB++;
				}
				else if(x==y){
					lowA++;
					lowB++;
					mycount++;
				}
			}
		}
		tid += gridDim.x*blockDim.x;
		
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();
}
__global__ void warp_merge_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/32;
	int i = threadIdx.x%32;
	int p = threadIdx.x/32;
	long int mycount=0;
	__shared__ index_t local[max_thd];
	__shared__ vertex_t A_diag[33*8];
	__shared__ vertex_t B_diag[33*8];	
	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
		if(i==0){
			A_diag[p*33+32]=m;
			B_diag[p*33+32]=n;
		}
		index_t index = (m+n)/32*i;
		vertex_t A_top, A_bottom, B_top, Ai, Bi;
		if(index>m){
			A_top = m;
			B_top = index-m;
		}
		else if(index<=m){
			A_top = index;
			B_top = 0;
		}
		if(index>n){
			A_bottom = index-n;
		}
		else if(index<=n){
			A_bottom = 0;
		}

		while(1){
			int offset=(A_top-A_bottom)/2;
			if(A_top==A_bottom){
				A_diag[p*33+i]=A_top;
				B_diag[p*33+i]=B_top;
				break;
			}

			Ai = A_top - offset;
			Bi = B_top + offset;
			if(offset<1){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_diag[p*33+i]=Ai-1;
					B_diag[p*33+i]=Bi+1;
					break;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}

			if(a[Ai]>b[Bi-1]){
				if(a[Ai-1]<b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi;
					break;
				}
				else if(a[Ai-1]>b[Bi]){
					A_top = Ai-1;
					B_top = Bi+1;
				}
				else if(a[Ai-1]==b[Bi]){
					A_diag[p*33+i]=Ai;
					B_diag[p*33+i]=Bi+1;
					break;
				}
			}
			else if(a[Ai]<b[Bi-1]){
				A_bottom = Ai+1;
			}
			else if(a[Ai]==b[Bi-1]){
				A_diag[p*33+i]=Ai+1;
				B_diag[p*33+i]=Bi;
				break;
			}
		}

//		__syncthreads();

		vertex_t lowA  = A_diag[p*33+i];
		vertex_t lowB  = B_diag[p*33+i];
		vertex_t highA = A_diag[p*33+i+1];
		vertex_t highB = B_diag[p*33+i+1];
		vertex_t x,y;
		while(lowA<highA && lowB<highB){
			x=a[lowA];
			y=b[lowB];
			if(x<y){
				lowA++;
			}
			else if(x>y){
				lowB++;
			}
			else if(x==y){
				lowA++;
				lowB++;
				mycount++;
			}
		}
//		tid += blockDim.x * gridDim.x/32;
		tid += gridDim.x*blockDim.x/32;
		
//		__syncthreads();
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();
}

__global__ void block_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = Ns + (threadIdx.x + blockIdx.x * blockDim.x)/ max_thd;
	int i = threadIdx.x% max_thd;
	index_t mycount=0;
//	__shared__ vertex_t cache[256];
	__shared__ index_t local[max_thd];

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];


		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
	
		local[i]=a[i*m/max_thd];	
		__syncthreads();

	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = max_thd;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[r];
				if(X==Y){
					mycount++;
					bot = top + max_thd;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/max_thd;
			top = top*m/max_thd -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += max_thd;
		
		}
		tid += gridDim.x*blockDim.x/ max_thd;
		__syncthreads();
	}

	//reduce
	__syncthreads();
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]+=val;
//		count[blockIdx.x]=val;
	}
}

__global__ void warp_binary_kernel
(	vertex_t*	head,
	vertex_t*	adj,
	vertex_t*	adj_list,
	index_t*	begin,
	index_t	Ns,
	index_t	Ne,
	index_t*	count
)
{
	//phase 1, partition
	index_t tid = (threadIdx.x + blockIdx.x * blockDim.x)/32 + Ns;
	index_t mycount=0;
	__shared__ index_t local[max_thd];

	int i = threadIdx.x%32;
	int p = threadIdx.x/32;

	while(tid<Ne){
		vertex_t A = head[tid];
		vertex_t B = adj[tid];
		index_t m = begin[A+1]-begin[A];//degree[A];
		index_t n = begin[B+1]-begin[B];//degree[B];

		index_t temp;	
		if(m<n){
			temp = A;
			A = B;
			B = temp;
			temp = m;
			m = n;
			n = temp;
		}

		vertex_t* a = &(adj_list[begin[A]]);
		vertex_t* b = &(adj_list[begin[B]]);
		
	//initial cache
		local[p*32+i]=a[i*m/32];	
		__syncthreads();
			
	//search
		int j=i;
		while(j<n){
			vertex_t X = b[j];
			vertex_t Y;
			//phase 1: cache
			int bot = 0;
			int top = 32;
			int r;
			while(top>bot+1){
				r = (top+bot)/2;
				Y = local[p*32+r];
				if(X==Y){
					mycount++;
					bot = top + 32;
				}
				if(X<Y){
					top = r;
				}
				if(X>Y){
					bot = r;
				}
			}
			//phase 2
			bot = bot*m/32;
			top = top*m/32 -1;
			while(top>=bot){
				r = (top+bot)/2;
				Y = a[r];
				if(X==Y){
					mycount++;
				}
				if(X<=Y){
					top = r-1;
				}
				if(X>=Y){
					bot = r+1;
				}
			}
			j += 32;
		
		}
		tid += blockDim.x*gridDim.x/32;
		__syncthreads();
	}

	__syncthreads();
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	if(threadIdx.x==0){
		index_t val=0;
		for(int i=0; i<blockDim.x; i++){
			val+= local[i];
		}
		count[blockIdx.x]=val;
	}
	__syncthreads();

}



__global__ void reduce_kernel2(index_t* count)
{
	for(int i=i; i<max_block; i++){
		count[i] += count[i-1];
	}
}

//---------------------------------------- cpu function--------------------
//------------------------------------------------------------------

void graph::scan(){

	hipSetDevice(4);
	vertex_t*	dev_adj;
	index_t*	dev_begin;
	index_t*	dev_count;

	H_ERR(hipMalloc(&dev_begin,  (vert_count+1)*sizeof(index_t)) );
	H_ERR(hipMalloc(&dev_count,    max_block*sizeof(index_t)) );
	
	index_t* block_offset;
	H_ERR(hipMalloc(&block_offset, max_block*sizeof(index_t)) );	
	H_ERR(hipMalloc(&dev_adj,  upperEdgeCount*sizeof(vertex_t)) );
	
	H_ERR(hipMemcpy(dev_adj,    upperAdj, upperEdgeCount*sizeof(vertex_t), hipMemcpyHostToDevice) );
	H_ERR(hipMemcpy(dev_begin,   upperBegin, (vert_count+1)*sizeof(index_t), hipMemcpyHostToDevice) );


double time1=wtime();


double time2=wtime();


	vertex_merge_kernel<<<max_block,max_thd>>>
	(	
		dev_adj,
		dev_begin,
		0,
		vert_count,
		dev_count
	);
	H_ERR(hipDeviceSynchronize() );
	
	reduce_kernel2 <<<1,1>>>(dev_count);
	H_ERR(hipDeviceSynchronize() );
	
	H_ERR(hipMemcpy(&count[0], dev_count, sizeof(index_t), hipMemcpyDeviceToHost));

double time4 = wtime();
	cout<<"total count = "<<count[0]<<endl;
	cout<<"GPU time = "<<time4-time2<<" seconds"<<endl;
	
	H_ERR(hipFree(dev_adj) );

	H_ERR(hipFree(dev_begin) );
	
	H_ERR(hipFree(dev_count) );
}


