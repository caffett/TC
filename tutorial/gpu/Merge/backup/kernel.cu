#include "hip/hip_runtime.h"
#include "comm.h"
#include "wtime.h"
#include "iostream"
#define max_thd 256 
#define max_block 256 

__global__ void tc_warp_kernel
(
	int*	adj_list,
	int*	head_list,
	int*	adj_card,
	int*	beg_pos,
	int*      count,
	int   vert_count,
	int   edge_count
){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//
	
	__shared__ int local[max_thd];	
	int mycount=0;
	int warp_id = tid/32;
	while(warp_id < edge_count){
		int U = head_list[warp_id];
		int V = adj_list[warp_id];
		int degreeU = adj_card[U];
		int degreeV = adj_card[V];
		int workId = threadIdx.x%32;
		int workload = degreeU * degreeV;
		//using a while loop to increase workId, to make a warp coordinate for one intersection
		while(workId < workload){
			int workU = workId/degreeV;
			int workV = workId%degreeV;
			int offsetU = beg_pos[U];
			int offsetV = beg_pos[V];
			if(adj_list[offsetU+workU] == adj_list[offsetV+workV]){
				mycount++;
			}
			workId+=32;
		}
		warp_id += blockDim.x * gridDim.x/32;
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	mycount=0;
	if(threadIdx.x==0){
		for(int i=0; i<max_thd; i++){
			mycount+= local[i];
		}
		count[blockIdx.x]=mycount;
	}
}

__global__ void tc_block_kernel
(
	int*	adj_list,
	int*	head_list,
	int*	adj_card,
	int*	beg_pos,
	int*      count,
	int   vert_count,
	int   edge_count
){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;//
	__shared__ int local[max_thd];	
	int mycount=0;
	int grid_id = blockIdx.x;
	while(grid_id < edge_count){
		int U = head_list[grid_id];
		int V = adj_list[grid_id];
		int degreeU = adj_card[U];
		int degreeV = adj_card[V];
		int offsetU = beg_pos[U];
		int offsetV = beg_pos[V];
		int workId = threadIdx.x;
		int workload = degreeU * degreeV;
		//using a while loop to increase workId, to make a warp coordinate for one intersection
		while(workId < workload){
			int workU = workId/degreeV;
			int workV = workId%degreeV;
			if(adj_list[offsetU+workU] == adj_list[offsetV+workV]){
				mycount++;
			}
			workId+=blockDim.x;
		}
		grid_id += gridDim.x;
	}
	//reduce
	local[threadIdx.x] = mycount;
	__syncthreads();
	mycount=0;
	if(threadIdx.x==0){
		for(int i=0; i<max_thd; i++){
			mycount+= local[i];
		}
		count[blockIdx.x]=mycount;
	}
	__syncthreads();
}

__global__ void reduce_kernel(int* count)
{
	int val = 0;
	for(int i=0; i<256; i++){
		val += count[i];
	}
	count[0] = val;
}



__global__ void tc_kernel
(
	int*	adj_list,
	int*	adj_card,
	int*	beg_pos,
	int*      count,
	int   vert_count,
	int   edge_count
){
 	__shared__ int thd_count[max_thd];
	int v = blockIdx.x;//vertex v
	int M = vert_count;

	
	int mycount=0;
	//step
	while(v<M){
		int local_count=0;
		int i = threadIdx.x;
		int N = adj_card[v];
		int beg = beg_pos[v];
		while(i<N){
			int w = adj_list[beg + i];//vertex w
			int s1 = beg_pos[v];	//start 1
			int e1 = beg_pos[v] + adj_card[v];	//end 1
			int s2 = beg_pos[w];
			int e2 = beg_pos[w] + adj_card[w];
/*			while(s1<e1 && s2<e2){
				if(adj_list[s1]<adj_list[s2]){
					s1++;
				}
				else if(adj_list[s1]>adj_list[s2]){
					s2++;
				}
				else if(adj_list[s1]==adj_list[s2]){
					s1++;
					s2++;
					local_count++;
				}
			}
*/
			for(int j=s1; j<e1; j++){
				for(int k=s2; k<e2; k++){
					if(adj_list[j]==adj_list[k]){
						local_count++;
						mycount++;
//						break;
					}
				}
			}

			i += blockDim.x;
		}
		thd_count[threadIdx.x]=local_count;
		//sycn
		__syncthreads();
		if(threadIdx.x==0){
			local_count=0;
			int k=0;
			for(k=0;k<max_thd;k++){
				local_count += thd_count[k];
			}
			count[v] = local_count;
		}
		v += gridDim.x;
		__syncthreads();
	}
} 



//template <typename vertex_t, typename index_t>
void graph//<vertex_t, index_t>
:: triangle_count()
{
	
	int*	dev_adj_list;
	int*	dev_head_list;
	int*	dev_adj_card;
	int*	dev_beg_pos;
	int*	dev_count;
	int*	dev_count2;

	hipMalloc(&dev_adj_list, upperEdgeCount*sizeof(int));
	hipMalloc(&dev_head_list, upperEdgeCount*sizeof(int));
	hipMalloc(&dev_adj_card, vert_count*sizeof(int));
	hipMalloc(&dev_beg_pos,  vert_count*sizeof(int));
	
	hipMalloc(&dev_count2,		vert_count*sizeof(int));
	hipMalloc(&dev_count,		max_thd*sizeof(int));

	hipMemcpy(dev_adj_list, upperAdj, upperEdgeCount*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_head_list, upperHead, upperEdgeCount*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_adj_card, upperDegree, vert_count*sizeof(int),  hipMemcpyHostToDevice);
	hipMemcpy(dev_beg_pos,  upperBegin,  vert_count*sizeof(int),  hipMemcpyHostToDevice);
	
	//
	hipDeviceSynchronize();
	tc_warp_kernel <<<max_block,max_thd>>>(
				dev_adj_list,
				dev_head_list,
				dev_adj_card,
				dev_beg_pos,
				dev_count,
				vert_count,
				upperEdgeCount
				);
	hipDeviceSynchronize();
	reduce_kernel <<<1,1>>>(dev_count);
//	tc_kernel <<<max_block,max_thd>>>(
//				dev_adj_list,
//				dev_adj_card,
//				dev_beg_pos,
//				dev_count2,
//				vert_count,
//				edge_count
//				);
//	hipMemcpy(valid, dev_count, vert_count*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(count, dev_count, sizeof(int), hipMemcpyDeviceToHost);

//	hipFree(dev_adj_list);
//	hipFree(dev_head_list);
//	hipFree(dev_adj_card);
//	hipFree(dev_beg_pos);
//	hipFree(dev_count);
	
}
