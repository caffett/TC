#include "hip/hip_runtime.h"
//sort.cu
#include "comm.h"
#include "graph.h"
#include "iostream"
#define T 56
using namespace std;


void printGraph(vertex_t vertCount, 
		vertex_t* head, 
		vertex_t* adj, 
		index_t* begin){
	for(vertex_t i=0; i<vertCount; i++){
		if(begin[i+1]>begin[i]){
			cout<<begin[i]<<" "<<begin[i+1]-begin[i]<<": ";
		}
//		for(int j=0; j<degree[i]; j++){
		for(vertex_t j=0; j<begin[i+1]-begin[i]; j++){
			cout<<head[begin[i]+j]<<"-"<<adj[begin[i]+j]<<" ";
		}
		if(begin[i+1]>begin[i]){
			cout<<"\n";
		}
	}
}

void quickSort(vertex_t * arr, index_t left, index_t right) {
      index_t i = left, j = right;
      vertex_t tmp;
      vertex_t pivot = arr[(left + right) / 2];
 
      /* partition */
      while (i <= j) {
            while (arr[i] < pivot)
                  i++;
            while (arr[j] > pivot)
                  j--;
            if (i <= j) {
                  tmp = arr[i];
                  arr[i] = arr[j];
                  arr[j] = tmp;
                  i++;
                  j--;
            }
      };
 
      /* recursion */
      if (left < j)
            quickSort(arr, left, j);
      if (i < right)
            quickSort(arr, i, right);
}


//sort and trim it to upper triangular
void graph::sort(){
	for(vertex_t i=0; i<vert_count; i++){
		index_t a=beg_pos[i];
		index_t b=beg_pos[i+1]-1;
		quickSort(adj_list,a,b);
	}
	
	upperBegin	= new index_t[vert_count+1];
	upperBegin[0]=0;
	index_t k=0;
	for(vertex_t i=0; i<vert_count; i++){
		upperBegin[i+1]=upperBegin[i];//upperDegree[i]=0;
		index_t j=beg_pos[i];
		while(j<beg_pos[i+1]){
			if(adj_list[j]==adj_list[j+1]&&head_list[j]==head_list[j+1])
			{
				j++;
				continue;
			}
			if(head_list[j]<adj_list[j]){
				k++;
				upperBegin[i+1]++;
			}
			j++;
		}
	}
	
	upperEdgeCount = k;
	upperAdj	= new vertex_t[upperEdgeCount];
	upperHead	= new vertex_t[upperEdgeCount];
//	upperDegree	= new index_t[vert_count];
//	int k=0;
	k=0;
	for(vertex_t i=0; i<vert_count; i++){
		index_t j=beg_pos[i];
		while(j<beg_pos[i+1]){
			if(adj_list[j]==adj_list[j+1]&&head_list[j]==head_list[j+1])
			{
				j++;
				continue;
			}
			if(head_list[j]<adj_list[j]){
				upperHead[k] =head_list[j];
				upperAdj[k] =adj_list[j];
				k++;
			}
			j++;
		}
	}
	
//	upperEdgeCount = k;
	cout<<"upper Edge Count= "<<upperEdgeCount<<"\n";
//	upperBegin[0] = 0;
//	for(int i=0; i<vert_count;i++){
//		upperBegin[i+1] += upperBegin[i];// + upperDegree[i-1];
//	}

}

void graph::reduce(){
	upperBegin	= new index_t[vert_count+1];
	upperBegin[0]=0;
	index_t k=0;
	for(vertex_t i=0; i<vert_count; i++){
		upperBegin[i+1]=upperBegin[i];//upperDegree[i]=0;
		index_t j=beg_pos[i];
		while(j<beg_pos[i+1]){
			if(head_list[j]<adj_list[j]){
				k++;
				upperBegin[i+1]++;//upperDegree[i]++;
			}
			j++;
		}
	}
	
	upperEdgeCount = k;
	upperAdj	= new vertex_t[upperEdgeCount];
	upperHead	= new vertex_t[upperEdgeCount];
	k=0;
	for(vertex_t i=0; i<vert_count; i++){
		index_t j=beg_pos[i];
		while(j<beg_pos[i+1]){
			if(head_list[j]<adj_list[j]){
				upperHead[k] =head_list[j];
				upperAdj[k] =head_list[j];
				k++;
			}
			j++;
		}
	}
	
	cout<<"upper Edge Count= "<<upperEdgeCount<<"\n";
}


void graph::reverse_rank_by_degree(){
	upperBegin	= new index_t[vert_count+1];
	upperBegin[0]=0;
	index_t k=0;
	for(vertex_t i=0; i<vert_count; i++){
		upperBegin[i+1]=upperBegin[i];//upperDegree[i]=0;
		index_t j=beg_pos[i];
			vertex_t h=head_list[j];
			index_t dh=beg_pos[h+1]-beg_pos[h];
		while(j<beg_pos[i+1]){
			vertex_t a=adj_list[j];
			index_t da=beg_pos[a+1]-beg_pos[a];
			if(dh>da||(dh==da && h<a)){
				k++;
				upperBegin[i+1]++;//upperDegree[i]++;
			}
			j++;
		}
	}
	
	upperEdgeCount = k;
	upperAdj	= new vertex_t[upperEdgeCount];
	upperHead	= new vertex_t[upperEdgeCount];
	k=0;
	for(vertex_t i=0; i<vert_count; i++){
		index_t j=beg_pos[i];
			vertex_t h=head_list[j];
			index_t dh=beg_pos[h+1]-beg_pos[h];
		while(j<beg_pos[i+1]){
			vertex_t a=adj_list[j];
			index_t da=beg_pos[a+1]-beg_pos[a];
			if(dh>da||(dh==da && h<a)){
				upperAdj[k] =adj_list[j];
				upperHead[k] =head_list[j];
				k++;
			}
			j++;
		}
	}
	
	cout<<"upper Edge Count= "<<upperEdgeCount<<"\n";
}

//rank-by-degree with trim
void graph::rank_by_degree(){
	upperBegin	= new index_t[vert_count+1];
	upperBegin[0]=0;
	#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for(vertex_t i=0; i<vert_count; i++){
//		upperBegin[i+1]=upperBegin[i];//upperDegree[i]=0;
		upperBegin[i+1]=0;
		index_t j=beg_pos[i];
		vertex_t h=head_list[j];
		index_t dh=beg_pos[h+1]-beg_pos[h];
		while(j<beg_pos[i+1]){
			vertex_t a=adj_list[j];
			index_t da=beg_pos[a+1]-beg_pos[a];
			if(dh<da || (dh==da && h<a)){
				upperBegin[i+1]++;//upperDegree[i]++;
			}
			j++;
		}
	}
	
	for(vertex_t i=0; i<vert_count; i++){
		upperBegin[i+1] += upperBegin[i];//upperDegree[i]=0;
	}

	upperEdgeCount = upperBegin[vert_count];//k;
	upperAdj	= new vertex_t[upperEdgeCount];
	upperHead	= new vertex_t[upperEdgeCount];
//#pragma omp parallel for num_threads(56) schedule(static)
#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for(vertex_t i=0; i<vert_count; i++){
		index_t j=beg_pos[i];
		index_t jj=upperBegin[i];
		vertex_t h=head_list[j];
		index_t dh=beg_pos[h+1]-beg_pos[h];
		while(j<beg_pos[i+1]){
			vertex_t a=adj_list[j];
			index_t da=beg_pos[a+1]-beg_pos[a];
			if(dh<da || (dh==da && h<a)){
				upperAdj[jj] =adj_list[j];
				upperHead[jj] =head_list[j];
				jj++;//k++;
			}
			j++;
		}
	}
	
	cout<<"upper Edge Count= "<<upperEdgeCount<<"\n";
}
/*function to search the begin position to find proper place to cut adjacent list
 return is the smallest position in the data array that value is equal or larger then lookup x
 */
vertex_t BinarySearch(vertex_t x, vertex_t*A, vertex_t bot, vertex_t top){

//	for(int i=bot;i<=top;i++){
//		cout<<A[i]<<" ";
//	}
//	cout<<"\n";

	vertex_t r= (bot+top)/2;
//	int result;
	while(top>bot){
		if(x<A[r]){
			top = r;
		}
		else if(x>A[r]){
			bot = r+1;
		}
		else if(x==A[r]){
			break;
		}
		r = (bot+top)/2;
	}
	return r;
}
vertex_t BinarySearch(index_t x, index_t*A, vertex_t bot, vertex_t top){

//	for(int i=bot;i<=top;i++){
//		cout<<A[i]<<" ";
//	}
//	cout<<"\n";

	vertex_t r= (bot+top)/2;
//	int result;
	while(top>bot){
		if(x<A[r]){
			top = r;
		}
		else if(x>A[r]){
			bot = r+1;
		}
		else if(x==A[r]){
			break;
		}
		r = (bot+top)/2;
	}
	return r;
}

void graph::partition(){
	//step 1, evenly cut the upper CSR by using binary search in upperBegin
	partAdj  = new vertex_t*[PART_NUM];
	partHead = new vertex_t*[PART_NUM];
//	partDegree = new index_t*[PART_NUM];
	partBegin  = new index_t*[PART_NUM];
	partEdgeCount = new index_t[PART_NUM];
	index_t offset[PART_NUM+1];
	offset[0] = 0;
	offset[PART_NUM] = upperEdgeCount;
	for(int i=1; i<PART_NUM; i++){
		index_t k=i*upperEdgeCount/PART_NUM;
		cout<<"k="<<k<<"\n";
		vertex_t index = BinarySearch(k, upperBegin, 0, vert_count-1);
		offset[i] = upperBegin[index];
		cout<<"part "<<i<<" cut at "<<index<<"\n";
	}
	vertex_t *destAdj = new vertex_t[upperEdgeCount];
	vertex_t *destHead= new vertex_t[upperEdgeCount];
//printGraph(vert_count, upperAdj, upperHead, upperBegin);
//	memcpy(destAdj,upperHead,upperEdgeCount*sizeof(int));
//	memcpy(destHead,upperAdj,upperEdgeCount*sizeof(int));
	//we are going to use space on origin int pointer upperAdj and upperHead as output
	//thus we need two new array to store input data.
	
	//step 2, exchange two end points of each edge, get the lower CSC
	for(int i=0;i<PART_NUM;i++){
		
		vertex_t* tempHead = &upperHead[offset[i]];
		vertex_t* tempAdj  = &upperAdj[offset[i]];
		partHead[i] = &destHead[offset[i]];
		partAdj[i]  = &destAdj[offset[i]];

		partEdgeCount[i] = offset[i+1]-offset[i];
cout<<"part "<<i<<" edge "<<partEdgeCount[i]<< "\n";

		partBegin[i] =  new index_t[vert_count+1];
//		partDegree[i] = new index_t[vert_count];
//		memset(partBegin[i],0,(vert_count+1)*sizeof(index_t));
		for(vertex_t j=0; j<vert_count+1; j++){
			partBegin[i][j]=0;
		}
		index_t *partOffset = new index_t[vert_count];
	//step 3, re-organize 1: go through new head list once to get new lowerDegree
		for(index_t j=0; j<partEdgeCount[i]; j++){
			//partDegree
			vertex_t head = tempHead[j];
			//partDegree[i][head]++; 
			partBegin[i][head+1]++;

		}

	//step 4, re-organize 2: prefix, input is degree, output is begin position
		partBegin[i][0]=0;
		for(vertex_t j=0; j<vert_count; j++){
//			partBegin[i][j] = partBegin[i][j-1] + partDegree[i][j-1];
			partBegin[i][j+1] += partBegin[i][j];
			partOffset[j]=0;
		}
//		memset(partOffset, 0, vert_count*sizeof(index_t));

	//step 5, re-organize 3: go through again and moving data to transfer CSC to CSR
		for(index_t j=0; j<partEdgeCount[i]; j++){
			vertex_t head = tempHead[j];
			partHead[i][partBegin[i][head]+partOffset[head]] = tempHead[j];
			partAdj [i][partBegin[i][head]+partOffset[head]] = tempAdj [j];
			partOffset[head]++;
		}
//		printGraph(vert_count, partHead[i], partAdj[i], partBegin[i]);

	}


}

void graph::preproc(){
	upperBegin	= new index_t[vert_count+1];
	upperBegin[0]=0;
	index_t k=0;
	index_t*inBegin = new index_t[vert_count+1];
#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for(vertex_t i=0; i<vert_count+1; i++){
		upperBegin[i]=0;
		inBegin[i]=0;

	}
//step 1: read round 1, to get the in-degree after orientation
#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for(vertex_t i=0; i<vert_count; i++){
//		upperBegin[i+1]=0;
		index_t j=beg_pos[i];
			vertex_t h=head_list[j];
			index_t dh=beg_pos[h+1]-beg_pos[h];
		while(j<beg_pos[i+1]){
			vertex_t a=adj_list[j];
			index_t da=beg_pos[a+1]-beg_pos[a];
			if(dh<da || (dh==da && h<a)){
//__sync_add_and_fetch(&k,1);
				upperBegin[i+1]++;//upperDegree[i]++;
				// to build rank-by-degree
//				__sync_fetch_and_add( &(inBegin[j+1]) , 1);
			}
			else{// if(dh>da || (dh==da && h>a)){
				inBegin[i+1]++;
			}
			j++;
		}
	}
	
	for(vertex_t i=0; i<vert_count; i++){
		upperBegin[i+1] += upperBegin[i];//upperDegree[i]=0;
		inBegin[i+1] += inBegin[i];//upperDegree[i]=0;
	}

	upperEdgeCount = upperBegin[vert_count];//k;
	upperAdj	= new vertex_t[upperEdgeCount];
	upperHead	= new vertex_t[upperEdgeCount];
//cout<<"test sycn_add_and_fetch k= "<<k<<endl;

//step 2: binary search in-degree for partition
	partAdj  = new vertex_t*[PART_NUM];
	partHead = new vertex_t*[PART_NUM];
	partBegin  = new index_t*[PART_NUM];
	partEdgeCount = new index_t[PART_NUM];
	index_t offset[PART_NUM+1];	// the vertex count begin from partition i
	vertex_t cutpoint[PART_NUM+1];	// the colum value begin from partition i
	offset[0] = 0;	
	offset[PART_NUM] = upperEdgeCount;
	cutpoint[0] = 0;
	cutpoint[PART_NUM] = vert_count+1;

//#pragma omp parallel for num_threads(PART_NUM) schedule(static)
	for(int i=1; i<PART_NUM; i++){
		index_t K=i*upperEdgeCount/PART_NUM;
		cout<<"K="<<K<<"\n";
		vertex_t index = BinarySearch(K, inBegin, 0, vert_count-1); // binary search
		cutpoint[i] = index; 			//used by each neigbhor list to find a place to cut	
		offset[i] = inBegin[index];
	}
//	vertex_t *destAdj = new vertex_t[upperEdgeCount];
//	vertex_t *destHead= new vertex_t[upperEdgeCount];

//#pragma omp parallel for num_threads(PART_NUM) schedule(static)
	for(int i=0;i<PART_NUM;i++){
		partBegin[i] =  new index_t[vert_count+1];
#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
		for(vertex_t n=0; n<vert_count+1; n++){
			partBegin[i][n]=0;
		}
	}


//step 3
#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for(vertex_t i=0; i<vert_count; i++){
		index_t j=beg_pos[i];
		index_t jj=upperBegin[i];
//		vertex_t h=head_list[j];
		vertex_t h=i;
		index_t dh=beg_pos[h+1]-beg_pos[h];
		//collect begin position for each partition
		vertex_t voffset[PART_NUM+1];
		voffset[0] = 0;
		voffset[PART_NUM]=dh;
//cout<<"for neighbors of "<<i<<endl;
		for(int n=0; n<PART_NUM; n++){
//			voffset[n]=BinarySearch(cutpoint[n], &adj_list[j], 0, dh-1);
			voffset[n]=BinarySearch(cutpoint[n], &adj_list[j], 0, dh);
//cout<<"part "<<n<<" start from voffset "<<voffset[n]<<" "<<adj_list[j+voffset[n]]<<endl;
		}
//		for(int n=0; n<PART_NUM; n++){
//			partBegin[n][i+1] = voffset[n+1] - voffset[n];
//		}
		//build oriented graph
//		/*
		for(int n=0; n<PART_NUM; n++){
			for(int nn=voffset[n]; nn<voffset[n+1]; nn++ ){
				vertex_t a = adj_list[j+nn];
				index_t da=beg_pos[a+1]-beg_pos[a];
//cout<<"check for edge "<<i<<"-"<<a<<endl;
				if(dh<da || (dh==da && h<a)){
					partBegin[n][i+1]++;
					upperAdj[jj]  = adj_list[j+nn];
					upperHead[jj] = i;//head_list[j+nn];
					jj++;//k++;
//cout<<"partition "<<n<<" keep edge "<<i<<"-"<<a<<endl;
				}
			}

		}
		
//		*/
		/*
		while(j<beg_pos[i+1]){
			vertex_t a=adj_list[j];
			index_t da=beg_pos[a+1]-beg_pos[a];
			if(dh<da || (dh==da && h<a)){
				upperAdj[jj] =adj_list[j];
				upperHead[jj] =head_list[j];
				jj++;//k++;
			}
			j++;
		}
		*/
	}
	
//#pragma omp parallel for num_threads(PART_NUM) schedule(static)
	for(int i=0;i<PART_NUM;i++){
		for(vertex_t j=0; j<vert_count; j++){
			partBegin[i][j+1] += partBegin[i][j];//upperDegree[i]=0;
		}
	}

	for(int i=0;i<PART_NUM;i++){
		partEdgeCount[i] = partBegin[i][vert_count];		//set the edge number of each partition
//		cout<<"part "<<i<<" edge  "<<partEdgeCount[i]<< "\n";
		partAdj[i]   =  new vertex_t[partEdgeCount[i]];		//allocate space for each partition
//		cout<<"part "<<i<<" edge~ "<<offset[i+1]-offset[i]<< "\n";
	}

	
//	for(int i=0;i<PART_NUM;i++){
//		cout<<"part "<<i<<" cut at "<<cutpoint[i]<<"\n";
//	}
//step 4: moving partition data	
#pragma omp parallel for num_threads(56) schedule(dynamic,1024)
	for(vertex_t i=0; i<vert_count; i++){
		index_t j=beg_pos[i];
		index_t jj=upperBegin[i];
		vertex_t h=head_list[j];
		index_t dh=beg_pos[h+1]-beg_pos[h];
		//collect begin position for each partition
		vertex_t voffset[5];
		voffset[0] = 0;
		voffset[PART_NUM]=dh;
		for(int n=0; n<PART_NUM; n++){
//			voffset[n]=BinarySearch(cutpoint[n], &adj_list[j], 0, dh-1);
			voffset[n]=BinarySearch(cutpoint[n], &adj_list[j], 0, dh);
		}
//		for(int n=0; n<PART_NUM; n++){
//			partBegin[n][i+1] = voffset[n+1] - voffset[n];
//		}
		//build oriented graph
		for(int n=0; n<PART_NUM; n++){
			index_t nnn=partBegin[n][i];
			for(int nn=voffset[n]; nn<voffset[n+1]; nn++ ){
				vertex_t a = adj_list[j+nn];
				index_t da=beg_pos[a+1]-beg_pos[a];
				if(dh<da || (dh==da && h<a)){
					partAdj[n][nnn] = a;
					nnn++;	
				}
			}

		}

	}
	cout<<"upper Edge Count= "<<upperEdgeCount<<"\n";
}
